#include "hip/hip_runtime.h"
﻿#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <>
#include <opencv2\opencv_modules.hpp>
#include <iostream>
#include <time.h>
using namespace std;

// 定义测试矩阵的维度
int const M = 5;
int const N = 10;

extern "C"
void addWithCuda(int *c, const int *a, const int *b, unsigned int size);
extern "C"
void getDeviceInfo();
__global__ void addKernel(int *c, const int *a, const int *b)
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}
int MatMultiply();

//int main()
//{
//    getDeviceInfo();
//    MatMultiply();
//    std::cout << std::endl << std::endl;

//    const int arraySize = 5;
//    const int a[arraySize] = { 1, 2, 3, 4, 5 };
//    const int b[arraySize] = { 10, 20, 30, 40, 50 };
//    int c[arraySize] = { 0 };

//    // Add vectors in parallel.
//    hipError_t cudaStatus = addWithCuda(c, a, b, arraySize);
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "addWithCuda failed!");
//        return 1;
//    }

//    printf("{1,2,3,4,5} + {10,20,30,40,50} = {%d,%d,%d,%d,%d}\n",
//        c[0], c[1], c[2], c[3], c[4]);

//    // hipDeviceReset must be called before exiting in order for profiling and
//    // tracing tools such as Nsight and Visual Profiler to show complete traces.
//    cudaStatus = hipDeviceReset();
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipDeviceReset failed!");
//        return 1;
//    }
//    return 0;
//}

extern "C"
void getDeviceInfo()
{
    int deviceCount;
    hipGetDeviceCount(&deviceCount);

    int dev;
    for (dev = 0; dev < deviceCount; dev++)
    {
        int driver_version(0), runtime_version(0);
        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, dev);
        if (dev == 0)
            if (deviceProp.minor = 9999 && deviceProp.major == 9999)
                printf("\n");
        printf("\nDevice%d:\"%s\"\n", dev, deviceProp.name);
        hipDriverGetVersion(&driver_version);
        printf("CUDA驱动版本:                                   %d.%d\n", driver_version / 1000, (driver_version % 1000) / 10);
        hipRuntimeGetVersion(&runtime_version);
        printf("CUDA运行时版本:                                 %d.%d\n", runtime_version / 1000, (runtime_version % 1000) / 10);
        printf("设备计算能力:                                   %d.%d\n", deviceProp.major, deviceProp.minor);
        printf("Total amount of Global Memory:                  %zu bytes\n", deviceProp.totalGlobalMem);
        printf("Number of SMs:                                  %d\n", deviceProp.multiProcessorCount);
        printf("Total amount of Constant Memory:                %zu bytes\n", deviceProp.totalConstMem);
        printf("Total amount of Shared Memory per block:        %zu bytes\n", deviceProp.sharedMemPerBlock);
        printf("Total number of registers available per block:  %d\n", deviceProp.regsPerBlock);
        printf("Warp size:                                      %d\n", deviceProp.warpSize);
        printf("Maximum number of threads per SM:               %d\n", deviceProp.maxThreadsPerMultiProcessor);
        printf("Maximum number of threads per block:            %d\n", deviceProp.maxThreadsPerBlock);
        printf("Maximum size of each dimension of a block:      %d x %d x %d\n", deviceProp.maxThreadsDim[0],
            deviceProp.maxThreadsDim[1],
            deviceProp.maxThreadsDim[2]);
        printf("Maximum size of each dimension of a grid:       %d x %d x %d\n", deviceProp.maxGridSize[0], deviceProp.maxGridSize[1], deviceProp.maxGridSize[2]);
        printf("Maximum memory pitch:                           %zu bytes\n", deviceProp.memPitch);
        printf("Texture alignmemt:                              %zu bytes\n", deviceProp.texturePitchAlignment);
        printf("Clock rate:                                     %.2f GHz\n", deviceProp.clockRate * 1e-6f);
        printf("Memory Clock rate:                              %.0f MHz\n", deviceProp.memoryClockRate * 1e-3f);
        printf("Memory Bus Width:                               %d-bit\n", deviceProp.memoryBusWidth);
    }

}

// Helper function for using CUDA to add vectors in parallel.
void addWithCuda(int *c, const int *a, const int *b, unsigned int size)
{
    int *dev_a = 0;
    int *dev_b = 0;
    int *dev_c = 0;
    hipError_t cudaStatus;


    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
    addKernel << <1, size >> > (dev_c, dev_a, dev_b);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }

    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);

}

// 矩阵乘法
int MatMultiply()
{
    // 定义状态变量
    hipblasStatus_t status;

    // 在 内存 中为将要计算的矩阵开辟空间
    float *h_A = (float*)malloc(N*M * sizeof(float));
    float *h_B = (float*)malloc(N*M * sizeof(float));

    // 在 内存 中为将要存放运算结果的矩阵开辟空间
    float *h_C = (float*)malloc(M*M * sizeof(float));

    srand((unsigned int)time(0));
    // 为待运算矩阵的元素赋予 0-10 范围内的随机数
    for (int i = 0; i < N*M; i++) {
        h_A[i] = (float)(rand() % 10 + 1);
        h_B[i] = (float)(rand() % 10 + 1);

    }

    // 打印待测试的矩阵
    std::cout << "矩阵 A :" << std::endl;
    for (int i = 0; i < N*M; i++) {
        cout << h_A[i] << " ";
        if ((i + 1) % N == 0) cout << endl;
    }
    cout << endl;
    cout << "矩阵 B :" << endl;
    for (int i = 0; i < N*M; i++) {
        cout << h_B[i] << " ";
        if ((i + 1) % M == 0) cout << endl;
    }
    cout << endl;

    /*
    ** GPU 计算矩阵相乘
    */

    // 创建并初始化 CUBLAS 库对象
    hipblasHandle_t handle;
    status = hipblasCreate(&handle);

    if (status != HIPBLAS_STATUS_SUCCESS)
    {
        if (status == HIPBLAS_STATUS_NOT_INITIALIZED) {
            cout << "CUBLAS 对象实例化出错" << endl;
        }
        getchar();
        return EXIT_FAILURE;
    }

    float *d_A, *d_B, *d_C;
    // 在 显存 中为将要计算的矩阵开辟空间
    hipMalloc(
        (void**)&d_A,    // 指向开辟的空间的指针
        N*M * sizeof(float)    //　需要开辟空间的字节数
    );
    hipMalloc(
        (void**)&d_B,
        N*M * sizeof(float)
    );

    // 在 显存 中为将要存放运算结果的矩阵开辟空间
    hipMalloc(
        (void**)&d_C,
        M*M * sizeof(float)
    );

    // 将矩阵数据传递进 显存 中已经开辟好了的空间
    hipblasSetVector(
        N*M,    // 要存入显存的元素个数
        sizeof(float),    // 每个元素大小
        h_A,    // 主机端起始地址
        1,    // 连续元素之间的存储间隔
        d_A,    // GPU 端起始地址
        1    // 连续元素之间的存储间隔
    );
    hipblasSetVector(
        N*M,
        sizeof(float),
        h_B,
        1,
        d_B,
        1
    );

    // 同步函数
    hipDeviceSynchronize();

    // 传递进矩阵相乘函数中的参数，具体含义请参考函数手册。
    float a = 1; float b = 0;
    // 矩阵相乘。该函数必然将数组解析成列优先数组
    hipblasSgemm(
        handle,    // blas 库对象
        HIPBLAS_OP_T,    // 矩阵 A 属性参数
        HIPBLAS_OP_T,    // 矩阵 B 属性参数
        M,    // A, C 的行数
        M,    // B, C 的列数
        N,    // A 的列数和 B 的行数
        &a,    // 运算式的 α 值
        d_A,    // A 在显存中的地址
        N,    // lda
        d_B,    // B 在显存中的地址
        M,    // ldb
        &b,    // 运算式的 β 值
        d_C,    // C 在显存中的地址(结果矩阵)
        M    // ldc
    );

    // 同步函数
    hipDeviceSynchronize();

    // 从 显存 中取出运算结果至 内存中去
    hipblasGetVector(
        M*M,    //  要取出元素的个数
        sizeof(float),    // 每个元素大小
        d_C,    // GPU 端起始地址
        1,    // 连续元素之间的存储间隔
        h_C,    // 主机端起始地址
        1    // 连续元素之间的存储间隔
    );

    // 打印运算结果
    cout << "计算结果的转置 ( (A*B)的转置 )：" << endl;

    for (int i = 0; i < M*M; i++) {
        cout << h_C[i] << " ";
        if ((i + 1) % M == 0) cout << endl;
    }

    // 清理掉使用过的内存
    free(h_A);
    free(h_B);
    free(h_C);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    // 释放 CUBLAS 库对象
    hipblasDestroy(handle);

//    getchar();
    return 1;
}
